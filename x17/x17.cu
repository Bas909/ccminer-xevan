#include "hip/hip_runtime.h"
/**
 * X17 algorithm (X15 + sha512 + haval256)
 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"

#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

#define NBN 2

// Memory for the hash functions
static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

extern void x13_hamsi_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x15_whirlpool_cpu_free(int thr_id);

extern void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);

extern void x17_haval256_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t* resNonce, uint64_t target);
extern void bmw256_cpu_hash_32_full(int thr_id, uint32_t threads, uint32_t *g_hash);
extern void quark_bmw512_cpu_hash_64x(int thr_id, uint32_t threads, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_groestl512(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void groestl512_cpu_init(int thr_id, uint32_t threads);
extern void groestl512_cpu_hash(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_skein512(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void keccak_xevan_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash);
extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void xevan_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_sha512_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash);
extern void xevan_haval512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_nonceVector, uint32_t *d_outputHash);
extern void xevan_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void xevan_haval512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *resNonce, uint64_t target);
extern void xevan_groestl512_cpu_hash(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void keccak_xevan_cpu_hash_64_A(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_nonceVector, uint32_t *d_outputHash);
extern void quark_blake512_cpu_hash_128(int thr_id, uint32_t threads, uint32_t *d_outputHash);
extern void quark_groestl512_cpu_hash_128(int thr_id, uint32_t threads,  uint32_t *d_hash);
extern void x11_luffa512_cpu_hash_128(int thr_id, uint32_t threads,uint32_t *d_hash);



// X17 CPU Hash (Validation)
extern "C" void x17hash(void *output, const void *input)
{
	uint32_t _ALIGN(64) hash[32]; // 128 bytes required
	const int dataLen = 128;
//return;
	sph_blake512_context     ctx_blake;
	sph_bmw512_context       ctx_bmw;
	sph_groestl512_context   ctx_groestl;
	sph_skein512_context     ctx_skein;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_luffa512_context     ctx_luffa;
	sph_cubehash512_context  ctx_cubehash;
	sph_shavite512_context   ctx_shavite;
	sph_simd512_context      ctx_simd;
	sph_echo512_context      ctx_echo;
	sph_hamsi512_context     ctx_hamsi;
	sph_fugue512_context     ctx_fugue;
	sph_shabal512_context    ctx_shabal;
	sph_whirlpool_context    ctx_whirlpool;
	sph_sha512_context       ctx_sha512;
	sph_haval256_5_context   ctx_haval;

//print_hash(input,20);
	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);
//print_hash(hash,32);
	memset(&hash[16], 0, 64);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, dataLen);
	sph_bmw512_close(&ctx_bmw, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hash, dataLen);
	sph_groestl512_close(&ctx_groestl, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, dataLen);
	sph_skein512_close(&ctx_skein, hash);

//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, hash, dataLen);
	sph_jh512_close(&ctx_jh, hash);
//print_hash(hash,32);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hash, dataLen);
	sph_keccak512_close(&ctx_keccak, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hash, dataLen);
	sph_luffa512_close(&ctx_luffa, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hash, dataLen);
	sph_cubehash512_close(&ctx_cubehash, hash);
//print_hash(hash,32);
	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hash, dataLen);
	sph_shavite512_close(&ctx_shavite, hash);
//print_hash(hash,32);
	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hash, dataLen);
	sph_simd512_close(&ctx_simd, hash);
//print_hash(hash,32);
	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hash, dataLen);
	sph_echo512_close(&ctx_echo, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hash, dataLen);
	sph_hamsi512_close(&ctx_hamsi, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hash, dataLen);
	sph_fugue512_close(&ctx_fugue, hash);
//print_hash(hash,32);
	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, dataLen);
	sph_shabal512_close(&ctx_shabal, hash);
//print_hash(hash,32);
	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, hash, dataLen);
	sph_whirlpool_close(&ctx_whirlpool, hash);
//print_hash(hash,32);
//for(int i=0;i<32;i++)hash[i]=0;
	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512,(const void*) hash, dataLen);
	sph_sha512_close(&ctx_sha512,(void*) hash);
//print_hash(hash,32);
	sph_haval256_5_init(&ctx_haval);
	sph_haval256_5(&ctx_haval,(const void*) hash, dataLen);
	sph_haval256_5_close(&ctx_haval, hash);
//print_hash(hash,32);

	memset(&hash[8], 0, dataLen - 32);

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, hash, dataLen);
	sph_blake512_close(&ctx_blake, hash);

//print_hash(hash,32);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, dataLen);
	sph_bmw512_close(&ctx_bmw, hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hash, dataLen);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, dataLen);
	sph_skein512_close(&ctx_skein, hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, hash, dataLen);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hash, dataLen);
	sph_keccak512_close(&ctx_keccak, hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hash, dataLen);
	sph_luffa512_close(&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hash, dataLen);
	sph_cubehash512_close(&ctx_cubehash, hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hash, dataLen);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hash, dataLen);
	sph_simd512_close(&ctx_simd, hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hash, dataLen);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hash, dataLen);
	sph_hamsi512_close(&ctx_hamsi, hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hash, dataLen);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, dataLen);
	sph_shabal512_close(&ctx_shabal, hash);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, hash, dataLen);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512,(const void*) hash, dataLen);
	sph_sha512_close(&ctx_sha512,(void*) hash);

//print_hash(hash,32);
	sph_haval256_5_init(&ctx_haval);
	sph_haval256_5(&ctx_haval,(const void*) hash, dataLen);
	sph_haval256_5_close(&ctx_haval, hash);
//print_hash(hash,8);
	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };


void print_hash(unsigned int *data,int size){
for(int i=0;i<size;i++)
        gpulog(LOG_WARNING, 0,"%x ",data[i]);
gpulog(LOG_WARNING, 0,"-------------");
}


extern "C" int scanhash_x17(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done){

	int dev_id = device_map[thr_id];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
/*
	uint32_t default_throughput = 1<<20;
	
	if (strstr(device_name[dev_id], "GTX 970")) default_throughput+=256*256*6;
	if (strstr(device_name[dev_id], "GTX 980")) default_throughput =1<<22;
	
	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput); // 19=256*256*8;
*/
	uint32_t default_throughput;
	if(device_sm[dev_id]<=500) default_throughput = 1<<20;
	else if(device_sm[dev_id]<=520) default_throughput = 1<<21;
	else if(device_sm[dev_id]>520) default_throughput = (1<<22) + (1<<21);
	default_throughput = 1<<20;
	if((strstr(device_name[dev_id], "1070")))default_throughput = 1<<20;
	if((strstr(device_name[dev_id], "1080")))default_throughput = 1<<20;
	
	uint32_t throughput = cuda_default_throughput(thr_id, default_throughput); // 19=256*256*8;
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	throughput&=0xFFFFFF70; //multiples of 128 due to simd_echo kernel

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0xff;

	gpulog(LOG_INFO,thr_id,"target %x %x %x",ptarget[5], ptarget[6], ptarget[7]);
        gpulog(LOG_INFO,thr_id,"target %llx",*(uint64_t*)&ptarget[6]);

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
//			hipDeviceSetCacheConfig(hipFuncCachePreferShared);
		}
		gpulog(LOG_INFO,thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

//		x11_simd_echo_512_cpu_init(thr_id, throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 0);
		groestl512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
//for(;;);
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint64_t) * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		h_resNonce[thr_id] = (uint32_t*) malloc(NBN  * 8 * sizeof(uint32_t));
		if(h_resNonce[thr_id] == NULL){
			gpulog(LOG_ERR,thr_id,"Host memory allocation failed");
			exit(EXIT_FAILURE);
		}		
		init[thr_id] = true;
	}

	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);
//		endiandata[k]=0;
//	print_hash(endiandata,20);
	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));
//	x11_simd512_cpu_init(thr_id, throughput);
//	for(;;);
	do {
		// Hash with CUDA


		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);//A
		quark_groestl512_cpu_hash_128(thr_id, throughput, d_hash[thr_id]);

		quark_skein512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);
		quark_jh512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);//A //fast
//		keccak_xevan_cpu_hash_64_A(thr_id, throughput,  d_hash[thr_id]);//A

//hipMemset(d_hash[thr_id], 0x00, 16*sizeof(uint32_t));
//		x11_luffa512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //P
//hipMemcpy(h_resNonce[thr_id], &d_hash[thr_id][0], 16*sizeof(uint32_t), hipMemcpyDeviceToHost);
//print_hash(h_resNonce[thr_id],16);
//hipMemset(d_hash[thr_id], 0x00, 16*sizeof(uint32_t));

		x11_luffa512_cpu_hash_128(thr_id, throughput, d_hash[thr_id]);//A
//hipMemcpy(h_resNonce[thr_id], &d_hash[thr_id][0], 16*sizeof(uint32_t), hipMemcpyDeviceToHost);
//print_hash(h_resNonce[thr_id],16);
//for(;;);

		x11_cubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //A 256
		xevan_shavite512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);//P slow r2
                x11_simd512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);  //A slow r3

//                hipMemset(d_hash[thr_id], 0x00, 16*sizeof(uint32_t));


//		xevan_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //slow r1

//                hipMemcpy(h_resNonce[thr_id], &d_hash[thr_id][0], 16*sizeof(uint32_t), hipMemcpyDeviceToHost);
//		print_hash(h_resNonce[thr_id],16);


  //              hipMemset(d_hash[thr_id], 0x00, 16*sizeof(uint32_t));

		x11_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);//A


//                hipMemcpy(h_resNonce[thr_id], &d_hash[thr_id][0], 16*sizeof(uint32_t), hipMemcpyDeviceToHost);
 //               print_hash(h_resNonce[thr_id],16);

//for(;;);

                x13_hamsi512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //fast
		x13_fugue512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //fast ++
		x14_shabal512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //fast
		xevan_whirlpool_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //opt2
		xevan_sha512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //fast
		xevan_haval512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); //fast


//		xevan_blake512_cpu_hash_64(thr_id, throughput,  d_hash[thr_id]);//BAD
quark_blake512_cpu_hash_128(thr_id, throughput,  d_hash[thr_id]);//BAD

//
                quark_bmw512_cpu_hash_64x(thr_id, throughput, NULL, d_hash[thr_id]);
//                xevan_groestl512_cpu_hash(thr_id, throughput, d_hash[thr_id]);
quark_groestl512_cpu_hash_128(thr_id, throughput, d_hash[thr_id]);

//                xevan_skein512(thr_id, throughput, d_hash[thr_id]);
                quark_skein512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);

                quark_jh512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]);
//                keccak_xevan_cpu_hash_64_A(thr_id, throughput,  d_hash[thr_id]);
//                x11_luffa512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                x11_luffa512_cpu_hash_128(thr_id, throughput, d_hash[thr_id]);//A

                x11_cubehash512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                xevan_shavite512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);//move to shared
                x11_simd512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); 

//                xevan_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                x11_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);

                x13_hamsi512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                x13_fugue512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                x14_shabal512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                xevan_whirlpool_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);
                xevan_sha512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]);







/*
for(int i = 10000;i< 10016;i++){
                hipMemcpy(h_resNonce[thr_id], &d_hash[thr_id][16*i], 16*sizeof(uint32_t), hipMemcpyDeviceToHost);
print_hash(h_resNonce[thr_id],8);
}
		for(;;);

*/
		xevan_haval512_cpu_hash_64_final(thr_id, throughput, d_hash[thr_id],d_resNonce[thr_id],*(uint64_t*)&ptarget[6]);

		hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost);

		if (h_resNonce[thr_id][0] != UINT32_MAX){
			const uint32_t Htarg = ptarget[7];
			const uint32_t startNounce = pdata[19];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], startNounce + h_resNonce[thr_id][0]);
			x17hash(vhash64, endiandata);
//			*hashes_done = pdata[19] - first_nonce + throughput + 1;
//			pdata[19] = startNounce + h_resNonce[thr_id][0];
			gpulog(LOG_WARNING, 0,"NONCE FOUND ");
//			return 1;
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput + 1;
				work_set_target_ratio(work, vhash64);
				pdata[19] = startNounce + h_resNonce[thr_id][0];
				if (h_resNonce[thr_id][1] != UINT32_MAX) {
					pdata[21] = startNounce+h_resNonce[thr_id][1];
					if(!opt_quiet)
						gpulog(LOG_BLUE,dev_id,"Found 2nd nonce: %08x", pdata[21]);
					be32enc(&endiandata[19], pdata[21]);
					x17hash(vhash64, endiandata);
					if (bn_hash_target_ratio(vhash64, ptarget) > work->shareratio[0]){
						work_set_target_ratio(work, vhash64);
						xchg(pdata[19],pdata[21]);
					}
					res++;
				}
				return res;
			}
			else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", h_resNonce[thr_id][0]);
				hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t));				
			}
		}

		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > (uint64_t)throughput + pdata[19]));

	*hashes_done = pdata[19] - first_nonce + 1;

	return 0;
}

// cleanup
extern "C" void free_x17(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	free(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);
	hipFree(d_hash[thr_id]);

	x11_simd_echo_512_cpu_free(thr_id);
	x15_whirlpool_cpu_free(thr_id);
	hipDeviceSynchronize();
	init[thr_id] = false;
}
